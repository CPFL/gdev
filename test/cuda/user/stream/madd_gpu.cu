#include <stdint.h>
#include <hip/hip_runtime.h>
__global__
void add(uint32_t *a, uint32_t *b, uint32_t *c, uint32_t n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < n && j < n) {
	    for (int k = 0; k < (8192 * 256); ++k) {
		int idx = i * n + j;
		c[idx] = a[idx] + b[idx];
	    }
    }
}
